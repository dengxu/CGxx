#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <memory>

#include "../CG.h"
#include "CGCUDABase.h"
#include "kernel.h"
#include "utils.h"

/// Class implementing parallel kernels with CUDA.
class CGCUDA : public CGCUDABase {
  struct MatrixCRSCUDA : MatrixCRS, MatrixCRSDataCUDA {
    MatrixCRSCUDA(const MatrixCOO &coo) : MatrixCRS(coo) {}
  };
  struct MatrixELLCUDA : MatrixELL, MatrixELLDataCUDA {
    MatrixELLCUDA(const MatrixCOO &coo) : MatrixELL(coo) {}
  };

  Device device;

  virtual void init(const char *matrixFile) override;

  virtual void convertToMatrixCRS() override {
    matrixCRS.reset(new MatrixCRSCUDA(*matrixCOO));
  }
  virtual void convertToMatrixELL() override {
    matrixELL.reset(new MatrixELLCUDA(*matrixCOO));
  }

  virtual void doTransferTo() override;
  virtual void doTransferFrom() override;

  virtual void cpy(Vector _dst, Vector _src) override;
  virtual void matvecKernel(Vector _x, Vector _y) override;
  virtual void axpyKernel(floatType a, Vector _x, Vector _y) override;
  virtual void xpayKernel(Vector _x, floatType a, Vector _y) override;
  virtual floatType vectorDotKernel(Vector _a, Vector _b) override;

  virtual void applyPreconditionerKernel(Vector _x, Vector _y) override;
};

void CGCUDA::init(const char *matrixFile) {
  // Set the device for initialization.
  checkError(hipSetDevice(0));

  CG::init(matrixFile);

  device.calculateLaunchConfiguration(N);
}

void CGCUDA::doTransferTo() {
  // Allocate memory on the device and transfer necessary data.
  size_t vectorSize = sizeof(floatType) * N;
  checkedMalloc(&device.k, vectorSize);
  checkedMalloc(&device.x, vectorSize);
  checkedMemcpyToDevice(device.k, k, vectorSize);
  checkedMemcpyToDevice(device.x, x, vectorSize);

  checkedMalloc(&device.p, vectorSize);
  checkedMalloc(&device.q, vectorSize);
  checkedMalloc(&device.r, vectorSize);

  switch (matrixFormat) {
  case MatrixFormatCRS: {
    size_t ptrSize = sizeof(int) * (N + 1);
    size_t indexSize = sizeof(int) * nz;
    size_t valueSize = sizeof(floatType) * nz;

    checkedMalloc(&device.matrixCRS.ptr, ptrSize);
    checkedMalloc(&device.matrixCRS.index, indexSize);
    checkedMalloc(&device.matrixCRS.value, valueSize);

    checkedMemcpyToDevice(device.matrixCRS.ptr, matrixCRS->ptr, ptrSize);
    checkedMemcpyToDevice(device.matrixCRS.index, matrixCRS->index, indexSize);
    checkedMemcpyToDevice(device.matrixCRS.value, matrixCRS->value, valueSize);
    break;
  }
  case MatrixFormatELL: {
    int elements = matrixELL->elements;
    size_t lengthSize = sizeof(int) * N;
    size_t indexSize = sizeof(int) * elements;
    size_t dataSize = sizeof(floatType) * elements;

    checkedMalloc(&device.matrixELL.length, lengthSize);
    checkedMalloc(&device.matrixELL.index, indexSize);
    checkedMalloc(&device.matrixELL.data, dataSize);

    checkedMemcpyToDevice(device.matrixELL.length, matrixELL->length,
                          lengthSize);
    checkedMemcpyToDevice(device.matrixELL.index, matrixELL->index, indexSize);
    checkedMemcpyToDevice(device.matrixELL.data, matrixELL->data, dataSize);
    break;
  }
  default:
    assert(0 && "Invalid matrix format!");
  }
  if (preconditioner != PreconditionerNone) {
    checkedMalloc(&device.z, vectorSize);

    switch (preconditioner) {
    case PreconditionerJacobi:
      checkedMalloc(&device.jacobi.C, vectorSize);
      checkedMemcpyToDevice(device.jacobi.C, jacobi->C, vectorSize);
      break;
    default:
      assert(0 && "Invalid preconditioner!");
    }
  }

  checkedMalloc(&device.tmp, sizeof(floatType) * Device::MaxBlocks);
}

void CGCUDA::doTransferFrom() {
  // Copy back solution and free memory on the device.
  checkedMemcpy(x, device.x, sizeof(floatType) * N, hipMemcpyDeviceToHost);

  checkedFree(device.k);
  checkedFree(device.x);

  checkedFree(device.p);
  checkedFree(device.q);
  checkedFree(device.r);

  switch (matrixFormat) {
  case MatrixFormatCRS: {
    checkedFree(device.matrixCRS.ptr);
    checkedFree(device.matrixCRS.index);
    checkedFree(device.matrixCRS.value);
    break;
  }
  case MatrixFormatELL: {
    checkedFree(device.matrixELL.length);
    checkedFree(device.matrixELL.index);
    checkedFree(device.matrixELL.data);
    break;
  }
  default:
    assert(0 && "Invalid matrix format!");
  }
  if (preconditioner != PreconditionerNone) {
    checkedFree(device.z);

    switch (preconditioner) {
    case PreconditionerJacobi: {
      checkedFree(device.jacobi.C);
      break;
    }
    default:
      assert(0 && "Invalid preconditioner!");
    }
  }

  checkedFree(device.tmp);
}

void CGCUDA::cpy(Vector _dst, Vector _src) {
  floatType *dst = device.getVector(_dst);
  floatType *src = device.getVector(_src);

  checkedMemcpy(dst, src, sizeof(floatType) * N, hipMemcpyDeviceToDevice);
}

void CGCUDA::matvecKernel(Vector _x, Vector _y) {
  floatType *x = device.getVector(_x);
  floatType *y = device.getVector(_y);

  switch (matrixFormat) {
  case MatrixFormatCRS:
    matvecKernelCRS<<<device.blocksMatvec, Device::Threads>>>(
        device.matrixCRS.ptr, device.matrixCRS.index, device.matrixCRS.value, x,
        y, N);
    break;
  case MatrixFormatELL:
    matvecKernelELL<<<device.blocksMatvec, Device::Threads>>>(
        device.matrixELL.length, device.matrixELL.index, device.matrixELL.data,
        x, y, N);
    break;
  default:
    assert(0 && "Invalid matrix format!");
  }
  checkLastError();
  checkedSynchronize();
}

void CGCUDA::axpyKernel(floatType a, Vector _x, Vector _y) {
  floatType *x = device.getVector(_x);
  floatType *y = device.getVector(_y);

  axpyKernelCUDA<<<device.blocks, Device::Threads>>>(a, x, y, N);
  checkLastError();
  checkedSynchronize();
}

void CGCUDA::xpayKernel(Vector _x, floatType a, Vector _y) {
  floatType *x = device.getVector(_x);
  floatType *y = device.getVector(_y);

  xpayKernelCUDA<<<device.blocks, Device::Threads>>>(x, a, y, N);
  checkLastError();
  checkedSynchronize();
}

floatType CGCUDA::vectorDotKernel(Vector _a, Vector _b) {
  floatType res = 0;
  floatType *a = device.getVector(_a);
  floatType *b = device.getVector(_b);

  // This is needed for warpReduceSum on __CUDA_ARCH__ < 350
  size_t sharedForVectorDot =
      max(Device::Threads, BlockReduction) * sizeof(floatType);
  size_t sharedForReduce =
      max(Device::MaxBlocks, BlockReduction) * sizeof(floatType);

  // https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
  vectorDotKernelCUDA<<<device.blocks, Device::Threads, sharedForVectorDot>>>(
      a, b, device.tmp, N);
  checkLastError();
  deviceReduceKernel<<<1, Device::MaxBlocks, sharedForReduce>>>(
      device.tmp, device.tmp, device.blocks);
  checkLastError();

  checkedMemcpy(&res, device.tmp, sizeof(floatType), hipMemcpyDeviceToHost);
  // The device is synchronized by the memory transfer.

  return res;
}

void CGCUDA::applyPreconditionerKernel(Vector _x, Vector _y) {
  floatType *x = device.getVector(_x);
  floatType *y = device.getVector(_y);

  switch (preconditioner) {
  case PreconditionerJacobi:
    applyPreconditionerKernelJacobi<<<device.blocks, Device::Threads>>>(
        device.jacobi.C, x, y, N);
    break;
  default:
    assert(0 && "Invalid preconditioner!");
  }
  checkLastError();
  checkedSynchronize();
}

CG *CG::getInstance() { return new CGCUDA; }
