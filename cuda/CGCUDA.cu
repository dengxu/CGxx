#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <memory>

#include "../CG.h"
#include "../Matrix.h"
#include "../Preconditioner.h"
#include "kernel.h"
#include "utils.h"

/// Class implementing parallel kernels with CUDA.
class CGCUDA : public CG {
  struct MatrixCRSCUDA : MatrixCRS {
    MatrixCRSCUDA(const MatrixCOO &coo) : MatrixCRS(coo) {}

    virtual void allocatePtr(int rows) override {
      checkedMallocHost(&ptr, sizeof(int) * (rows + 1));
    }
    virtual void deallocatePtr() override { checkedFreeHost(ptr); }
    virtual void allocateIndexAndValue(int values) override {
      checkedMallocHost(&index, sizeof(int) * values);
      checkedMallocHost(&value, sizeof(floatType) * values);
    }
    virtual void deallocateIndexAndValue() override {
      checkedFreeHost(index);
      checkedFreeHost(value);
    }
  };
  struct MatrixELLCUDA : MatrixELL {
    MatrixELLCUDA(const MatrixCOO &coo) : MatrixELL(coo) {}

    virtual void allocateLength(int rows) override {
      checkedMallocHost(&length, sizeof(int) * rows);
    }
    virtual void deallocateLength() override { checkedFreeHost(length); }
    virtual void allocateIndexAndData() override {
      checkedMallocHost(&index, sizeof(int) * elements);
      checkedMallocHost(&data, sizeof(floatType) * elements);
    }
    virtual void deallocateIndexAndData() override {
      checkedFreeHost(index);
      checkedFreeHost(data);
    }
  };
  struct JacobiCUDA : Jacobi {
    JacobiCUDA(const MatrixCOO &coo) : Jacobi(coo) {}

    virtual void allocateC(int N) override {
      checkedMallocHost(&C, sizeof(floatType) * N);
    }
    virtual void deallocateC() override { checkedFreeHost(C); }
  };

  const int Threads = 128;
  const int MaxBlocks = 1024;
  // 65536 seems to not work on the Pascal nodes.
  const int MaxBlocksMatvec = 65535;
  int blocks;
  int blocksMatvec;

  floatType *tmp;

  floatType *k_dev;
  floatType *x_dev;

  floatType *p_dev;
  floatType *q_dev;
  floatType *r_dev;
  floatType *z_dev;

  struct {
    int *ptr;
    int *index;
    floatType *value;
  } matrixCRS_dev;
  struct {
    int *length;
    int *index;
    floatType *data;
  } matrixELL_dev;
  struct {
    floatType *C;
  } jacobi_dev;

  floatType *getVector(Vector v) {
    switch (v) {
    case VectorK:
      return k_dev;
    case VectorX:
      return x_dev;
    case VectorP:
      return p_dev;
    case VectorQ:
      return q_dev;
    case VectorR:
      return r_dev;
    case VectorZ:
      return z_dev;
    }
    assert(0 && "Invalid value of v!");
    return nullptr;
  }

  virtual bool supportsMatrixFormat(MatrixFormat format) override {
    return format == MatrixFormatCRS || format == MatrixFormatELL;
  }
  virtual bool supportsPreconditioner(Preconditioner preconditioner) override {
    return preconditioner == PreconditionerJacobi;
  }

  virtual void init(const char *matrixFile) override;

  virtual void convertToMatrixCRS() override {
    matrixCRS.reset(new MatrixCRSCUDA(*matrixCOO));
  }
  virtual void convertToMatrixELL() override {
    matrixELL.reset(new MatrixELLCUDA(*matrixCOO));
  }

  virtual void initJacobi() override {
    jacobi.reset(new JacobiCUDA(*matrixCOO));
  }

  virtual void allocateK() override {
    checkedMallocHost(&k, sizeof(floatType) * N);
  }
  virtual void deallocateK() override { checkedFreeHost(k); }
  virtual void allocateX() override {
    checkedMallocHost(&x, sizeof(floatType) * N);
  }
  virtual void deallocateX() override { checkedFreeHost(x); }

  virtual bool needsTransfer() override { return true; }
  virtual void doTransferTo() override;
  virtual void doTransferFrom() override;

  virtual void cpy(Vector _dst, Vector _src) override;
  virtual void matvecKernel(Vector _x, Vector _y) override;
  virtual void axpyKernel(floatType a, Vector _x, Vector _y) override;
  virtual void xpayKernel(Vector _x, floatType a, Vector _y) override;
  virtual floatType vectorDotKernel(Vector _a, Vector _b) override;

  virtual void applyPreconditionerKernel(Vector _x, Vector _y) override;

public:
  CGCUDA() : CG(MatrixFormatELL, PreconditionerJacobi) {}
};

void CGCUDA::init(const char *matrixFile) {
  // Set the device for initialization.
  checkError(hipSetDevice(0));

  CG::init(matrixFile);

  blocks = calculateBlocks(N, Threads, MaxBlocks);
  blocksMatvec = calculateBlocks(N, Threads, MaxBlocksMatvec);
}

void CGCUDA::doTransferTo() {
  // Allocate memory on the device and transfer necessary data.
  size_t vectorSize = sizeof(floatType) * N;
  checkedMalloc(&k_dev, vectorSize);
  checkedMalloc(&x_dev, vectorSize);
  checkedMemcpyToDevice(k_dev, k, vectorSize);
  checkedMemcpyToDevice(x_dev, x, vectorSize);

  checkedMalloc(&p_dev, vectorSize);
  checkedMalloc(&q_dev, vectorSize);
  checkedMalloc(&r_dev, vectorSize);

  switch (matrixFormat) {
  case MatrixFormatCRS: {
    size_t ptrSize = sizeof(int) * (N + 1);
    size_t indexSize = sizeof(int) * nz;
    size_t valueSize = sizeof(floatType) * nz;

    checkedMalloc(&matrixCRS_dev.ptr, ptrSize);
    checkedMalloc(&matrixCRS_dev.index, indexSize);
    checkedMalloc(&matrixCRS_dev.value, valueSize);

    checkedMemcpyToDevice(matrixCRS_dev.ptr, matrixCRS->ptr, ptrSize);
    checkedMemcpyToDevice(matrixCRS_dev.index, matrixCRS->index, indexSize);
    checkedMemcpyToDevice(matrixCRS_dev.value, matrixCRS->value, valueSize);
    break;
  }
  case MatrixFormatELL: {
    int elements = matrixELL->elements;
    size_t lengthSize = sizeof(int) * N;
    size_t indexSize = sizeof(int) * elements;
    size_t dataSize = sizeof(floatType) * elements;

    checkedMalloc(&matrixELL_dev.length, lengthSize);
    checkedMalloc(&matrixELL_dev.index, indexSize);
    checkedMalloc(&matrixELL_dev.data, dataSize);

    checkedMemcpyToDevice(matrixELL_dev.length, matrixELL->length, lengthSize);
    checkedMemcpyToDevice(matrixELL_dev.index, matrixELL->index, indexSize);
    checkedMemcpyToDevice(matrixELL_dev.data, matrixELL->data, dataSize);
    break;
  }
  default:
    assert(0 && "Invalid matrix format!");
  }
  if (preconditioner != PreconditionerNone) {
    checkedMalloc(&z_dev, vectorSize);

    switch (preconditioner) {
    case PreconditionerJacobi:
      checkedMalloc(&jacobi_dev.C, vectorSize);
      checkedMemcpyToDevice(jacobi_dev.C, jacobi->C, vectorSize);
      break;
    default:
      assert(0 && "Invalid preconditioner!");
    }
  }

  checkedMalloc(&tmp, sizeof(floatType) * MaxBlocks);
}

void CGCUDA::doTransferFrom() {
  // Copy back solution and free memory on the device.
  checkedMemcpy(x, x_dev, sizeof(floatType) * N, hipMemcpyDeviceToHost);

  checkedFree(k_dev);
  checkedFree(x_dev);

  checkedFree(p_dev);
  checkedFree(q_dev);
  checkedFree(r_dev);

  switch (matrixFormat) {
  case MatrixFormatCRS: {
    checkedFree(matrixCRS_dev.ptr);
    checkedFree(matrixCRS_dev.index);
    checkedFree(matrixCRS_dev.value);
    break;
  }
  case MatrixFormatELL: {
    checkedFree(matrixELL_dev.length);
    checkedFree(matrixELL_dev.index);
    checkedFree(matrixELL_dev.data);
    break;
  }
  default:
    assert(0 && "Invalid matrix format!");
  }
  if (preconditioner != PreconditionerNone) {
    checkedFree(z_dev);

    switch (preconditioner) {
    case PreconditionerJacobi: {
      checkedFree(jacobi_dev.C);
      break;
    }
    default:
      assert(0 && "Invalid preconditioner!");
    }
  }

  checkedFree(tmp);
}

void CGCUDA::cpy(Vector _dst, Vector _src) {
  floatType *dst = getVector(_dst);
  floatType *src = getVector(_src);

  checkedMemcpy(dst, src, sizeof(floatType) * N, hipMemcpyDeviceToDevice);
}

void CGCUDA::matvecKernel(Vector _x, Vector _y) {
  floatType *x = getVector(_x);
  floatType *y = getVector(_y);

  switch (matrixFormat) {
  case MatrixFormatCRS:
    matvecKernelCRS<<<blocksMatvec, Threads>>>(
        matrixCRS_dev.ptr, matrixCRS_dev.index, matrixCRS_dev.value, x, y, N);
    break;
  case MatrixFormatELL:
    matvecKernelELL<<<blocksMatvec, Threads>>>(
        matrixELL_dev.length, matrixELL_dev.index, matrixELL_dev.data, x, y, N);
    break;
  default:
    assert(0 && "Invalid matrix format!");
  }
  checkLastError();
  checkedSynchronize();
}

void CGCUDA::axpyKernel(floatType a, Vector _x, Vector _y) {
  floatType *x = getVector(_x);
  floatType *y = getVector(_y);

  axpyKernelCUDA<<<blocks, Threads>>>(a, x, y, N);
  checkLastError();
  checkedSynchronize();
}

void CGCUDA::xpayKernel(Vector _x, floatType a, Vector _y) {
  floatType *x = getVector(_x);
  floatType *y = getVector(_y);

  xpayKernelCUDA<<<blocks, Threads>>>(x, a, y, N);
  checkLastError();
  checkedSynchronize();
}

floatType CGCUDA::vectorDotKernel(Vector _a, Vector _b) {
  floatType res = 0;
  floatType *a = getVector(_a);
  floatType *b = getVector(_b);

  // This is needed for warpReduceSum on __CUDA_ARCH__ < 350
  size_t sharedForVectorDot = max(Threads, BlockReduction) * sizeof(floatType);
  size_t sharedForReduce = max(MaxBlocks, BlockReduction) * sizeof(floatType);

  // https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
  vectorDotKernelCUDA<<<blocks, Threads, sharedForVectorDot>>>(a, b, tmp, N);
  checkLastError();
  deviceReduceKernel<<<1, MaxBlocks, sharedForReduce>>>(tmp, tmp, blocks);
  checkLastError();

  checkedMemcpy(&res, tmp, sizeof(floatType), hipMemcpyDeviceToHost);
  // The device is synchronized by the memory transfer.

  return res;
}

void CGCUDA::applyPreconditionerKernel(Vector _x, Vector _y) {
  floatType *x = getVector(_x);
  floatType *y = getVector(_y);

  switch (preconditioner) {
  case PreconditionerJacobi:
    applyPreconditionerKernelJacobi<<<blocks, Threads>>>(jacobi_dev.C, x, y, N);
    break;
  default:
    assert(0 && "Invalid preconditioner!");
  }
  checkLastError();
  checkedSynchronize();
}

CG *CG::getInstance() { return new CGCUDA; }
