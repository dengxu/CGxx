#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

#include "../CG.h"
#include "CGCUDABase.h"
#include "kernel.h"
#include "utils.h"

/// Class implementing parallel kernels with CUDA.
class CGMultiCUDA : public CGCUDABase {
  enum GatherImpl {
    GatherImplHost,
    GatherImplDevice,
    GatherImplP2P,
  };

  struct SplitMatrixCRSCUDA : SplitMatrixCRS {
    SplitMatrixCRSCUDA(const MatrixCOO &coo, const WorkDistribution &wd)
        : SplitMatrixCRS(coo, wd) {}

    virtual void allocateData(int numberOfChunks) override {
      data.reset((MatrixDataCRS *)new MatrixDataCRSCUDA[numberOfChunks]);
    }
  };
  struct SplitMatrixELLCUDA : SplitMatrixELL {
    SplitMatrixELLCUDA(const MatrixCOO &coo, const WorkDistribution &wd)
        : SplitMatrixELL(coo, wd) {}

    virtual void allocateData(int numberOfChunks) override {
      data.reset((MatrixDataELL *)new MatrixDataELLCUDA[numberOfChunks]);
    }
  };

  struct MultiDevice : Device {
    int id;
    CGMultiCUDA *cg;

    floatType vectorDotResult;
    hipStream_t gatherStream;

    ~MultiDevice() { checkError(hipStreamDestroy(gatherStream)); }

    void init(int id, CGMultiCUDA *cg) {
      this->id = id;
      this->cg = cg;

      setDevice();
      checkError(hipStreamCreate(&gatherStream));
    }
    void setDevice() const { checkedSetDevice(id); }

    floatType *getVector(Vector v) const override {
      return getVector(v, false);
    }
    floatType *getVector(Vector v, bool forMatvec) const {
      assert(!forMatvec || (v == VectorX || v == VectorP));

      floatType *res = Device::getVector(v);
      if (!forMatvec && (v == VectorX || v == VectorP)) {
        // These vectors are fully allocated, but we only need the "local" part.
        res += cg->workDistribution->offsets[id];
      }
      return res;
    }
  };

  std::vector<MultiDevice> devices;
  GatherImpl gatherImpl = GatherImplHost;

  floatType *p = nullptr;

  virtual int getNumberOfChunks() override { return devices.size(); }

  virtual void parseEnvironment() override;
  virtual void init(const char *matrixFile) override;

  virtual void convertToSplitMatrixCRS() {
    splitMatrixCRS.reset(new SplitMatrixCRSCUDA(*matrixCOO, *workDistribution));
  }
  virtual void convertToSplitMatrixELL() {
    splitMatrixELL.reset(new SplitMatrixELLCUDA(*matrixCOO, *workDistribution));
  }

  virtual void initJacobi() override {
    jacobi.reset(new JacobiCUDA(*matrixCOO));
  }

  virtual void allocateK() override {
    checkedMallocHost(&k, sizeof(floatType) * N);
  }
  virtual void deallocateK() override { checkedFreeHost(k); }
  virtual void allocateX() override {
    checkedMallocHost(&x, sizeof(floatType) * N);
  }
  virtual void deallocateX() override { checkedFreeHost(x); }

  void synchronizeAllDevices();

  void allocateAndCopyMatrixDataCRS(int length, const MatrixDataCRS &data,
                                    Device::MatrixCRSDevice &deviceMatrix);
  void allocateAndCopyMatrixDataELL(int length, const MatrixDataELL &data,
                                    Device::MatrixELLDevice &deviceMatrix);
  virtual void doTransferTo() override;
  void freeMatrixDataCRS(const Device::MatrixCRSDevice &device);
  void freeMatrixDataELL(const Device::MatrixELLDevice &device);
  virtual void doTransferFrom() override;

  virtual void cpy(Vector _dst, Vector _src) override;

  void matvecGatherXViaHost(Vector _x);
  void matvecGatherXOnDevices(Vector _x);
  virtual void matvecKernel(Vector _x, Vector _y) override;

  virtual void axpyKernel(floatType a, Vector _x, Vector _y) override;
  virtual void xpayKernel(Vector _x, floatType a, Vector _y) override;
  virtual floatType vectorDotKernel(Vector _a, Vector _b) override;

  virtual void applyPreconditionerKernel(Vector _x, Vector _y) override;

  virtual void printSummary() override;
  virtual void cleanup() override {
    CG::cleanup();

    if (gatherImpl == GatherImplHost) {
      checkedFreeHost(p);
    }
  }
};

const char *CG_CUDA_GATHER_IMPL = "CG_CUDA_GATHER_IMPL";
const char *CG_CUDA_GATHER_IMPL_HOST = "host";
const char *CG_CUDA_GATHER_IMPL_DEVICE = "device";
const char *CG_CUDA_GATHER_IMPL_P2P = "p2p";

void CGMultiCUDA::parseEnvironment() {
  CG::parseEnvironment();

  const char *env = std::getenv(CG_CUDA_GATHER_IMPL);
  if (env != NULL && *env != 0) {
    std::string lower(env);
    std::transform(lower.begin(), lower.end(), lower.begin(),
                   [](char c) { return std::tolower(c); });

    if (lower == CG_CUDA_GATHER_IMPL_HOST) {
      gatherImpl = GatherImplHost;
    } else if (lower == CG_CUDA_GATHER_IMPL_DEVICE) {
      gatherImpl = GatherImplDevice;
    } else if (lower == CG_CUDA_GATHER_IMPL_P2P) {
      gatherImpl = GatherImplP2P;
    } else {
      std::cerr << "Invalid value for " << CG_CUDA_GATHER_IMPL << "! ("
                << CG_CUDA_GATHER_IMPL_HOST << ", "
                << CG_CUDA_GATHER_IMPL_DEVICE << ", or "
                << CG_CUDA_GATHER_IMPL_P2P << ")" << std::endl;
      std::exit(1);
    }
  }
}

void CGMultiCUDA::init(const char *matrixFile) {
  int numberOfDevices;
  hipGetDeviceCount(&numberOfDevices);
  if (numberOfDevices < 2) {
    std::cerr << "Need at least 2 devices!" << std::endl;
    std::exit(1);
  }
  devices.resize(numberOfDevices);
  for (int d = 0; d < numberOfDevices; d++) {
    devices[d].init(d, this);
  }

  // Set each device once for initialization. Enable peer access if requested,
  // or abort if not available.
  // NOTE: hipDeviceEnablePeerAccess() is unidirectional, so it has to be
  // called once for each direction, twice per combination!
  for (MultiDevice &device : devices) {
    device.setDevice();

    if (gatherImpl == GatherImplP2P) {
      for (MultiDevice &peerDevice : devices) {
        if (peerDevice.id == device.id) {
          continue;
        }

        int canAccessPeer;
        checkError(
            hipDeviceCanAccessPeer(&canAccessPeer, device.id, peerDevice.id));
        if (!canAccessPeer) {
          std::cerr << "Device " << device.id << " cannot access "
                    << peerDevice.id << "!" << std::endl;
          std::exit(1);
        }

        checkError(hipDeviceEnablePeerAccess(peerDevice.id, 0));
      }
    }
  }

  CG::init(matrixFile);
  assert(workDistribution->numberOfChunks == numberOfDevices);

  for (int i = 0; i < numberOfDevices; i++) {
    int length = workDistribution->lengths[i];
    devices[i].calculateLaunchConfiguration(length);
  }

  if (gatherImpl == GatherImplHost) {
    checkedMallocHost(&p, sizeof(floatType) * N);
  }
}

void CGMultiCUDA::synchronizeAllDevices() {
  for (const MultiDevice &device : devices) {
    device.setDevice();
    checkedSynchronize();
  }
}

void CGMultiCUDA::allocateAndCopyMatrixDataCRS(
    int length, const MatrixDataCRS &data,
    Device::MatrixCRSDevice &deviceMatrix) {
  size_t ptrSize = sizeof(int) * (length + 1);
  int deviceNz = data.ptr[length];
  size_t indexSize = sizeof(int) * deviceNz;
  size_t valueSize = sizeof(floatType) * deviceNz;

  checkedMalloc(&deviceMatrix.ptr, ptrSize);
  checkedMalloc(&deviceMatrix.index, indexSize);
  checkedMalloc(&deviceMatrix.value, valueSize);

  checkedMemcpyAsyncToDevice(deviceMatrix.ptr, data.ptr, ptrSize);
  checkedMemcpyAsyncToDevice(deviceMatrix.index, data.index, indexSize);
  checkedMemcpyAsyncToDevice(deviceMatrix.value, data.value, valueSize);
}

void CGMultiCUDA::allocateAndCopyMatrixDataELL(
    int length, const MatrixDataELL &data,
    Device::MatrixELLDevice &deviceMatrix) {
  size_t lengthSize = sizeof(int) * length;
  int elements = data.elements;
  size_t indexSize = sizeof(int) * elements;
  size_t dataSize = sizeof(floatType) * elements;

  checkedMalloc(&deviceMatrix.length, lengthSize);
  checkedMalloc(&deviceMatrix.index, indexSize);
  checkedMalloc(&deviceMatrix.data, dataSize);

  checkedMemcpyAsyncToDevice(deviceMatrix.length, data.length, lengthSize);
  checkedMemcpyAsyncToDevice(deviceMatrix.index, data.index, indexSize);
  checkedMemcpyAsyncToDevice(deviceMatrix.data, data.data, dataSize);
}

void CGMultiCUDA::doTransferTo() {
  size_t fullVectorSize = sizeof(floatType) * N;

  // Allocate memory on all devices and transfer necessary data.
  for (MultiDevice &device : devices) {
    device.setDevice();

    int d = device.id;
    int offset = workDistribution->offsets[d];
    int length = workDistribution->lengths[d];

    size_t vectorSize = sizeof(floatType) * length;
    checkedMalloc(&device.k, vectorSize);
    checkedMalloc(&device.x, fullVectorSize);
    checkedMemcpyAsyncToDevice(device.k, k + offset, vectorSize);
    checkedMemcpyAsyncToDevice(device.x, x, fullVectorSize);

    checkedMalloc(&device.p, fullVectorSize);
    checkedMalloc(&device.q, vectorSize);
    checkedMalloc(&device.r, vectorSize);

    switch (matrixFormat) {
    case MatrixFormatCRS: {
      allocateAndCopyMatrixDataCRS(length, splitMatrixCRS->data[d],
                                   device.matrixCRS);
      break;
    }
    case MatrixFormatELL: {
      allocateAndCopyMatrixDataELL(length, splitMatrixELL->data[d],
                                   device.matrixELL);
      break;
    }
    default:
      assert(0 && "Invalid matrix format!");
    }
    if (preconditioner != PreconditionerNone) {
      checkedMalloc(&device.z, vectorSize);

      switch (preconditioner) {
      case PreconditionerJacobi:
        checkedMalloc(&device.jacobi.C, vectorSize);
        checkedMemcpyAsyncToDevice(device.jacobi.C, jacobi->C + offset,
                                   vectorSize);
        break;
      default:
        assert(0 && "Invalid preconditioner!");
      }
    }

    checkedMalloc(&device.tmp, sizeof(floatType) * Device::MaxBlocks);
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::freeMatrixDataCRS(const Device::MatrixCRSDevice &device) {
  checkedFree(device.ptr);
  checkedFree(device.index);
  checkedFree(device.value);
}

void CGMultiCUDA::freeMatrixDataELL(const Device::MatrixELLDevice &device) {
  checkedFree(device.length);
  checkedFree(device.index);
  checkedFree(device.data);
}

void CGMultiCUDA::doTransferFrom() {
  // Copy back solution and free memory on the device.
  for (MultiDevice &device : devices) {
    device.setDevice();

    int d = device.id;
    int offset = workDistribution->offsets[d];
    int length = workDistribution->lengths[d];

    checkedMemcpyAsync(x + offset, device.x + offset,
                       sizeof(floatType) * length, hipMemcpyDeviceToHost);

    checkedFree(device.k);
    checkedFree(device.x);

    checkedFree(device.p);
    checkedFree(device.q);
    checkedFree(device.r);

    switch (matrixFormat) {
    case MatrixFormatCRS: {
      freeMatrixDataCRS(device.matrixCRS);
      break;
    }
    case MatrixFormatELL: {
      freeMatrixDataELL(device.matrixELL);
      break;
    }
    default:
      assert(0 && "Invalid matrix format!");
    }
    if (preconditioner != PreconditionerNone) {
      checkedFree(device.z);

      switch (preconditioner) {
      case PreconditionerJacobi: {
        checkedFree(device.jacobi.C);
        break;
      }
      default:
        assert(0 && "Invalid preconditioner!");
      }
    }

    checkedFree(device.tmp);
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::cpy(Vector _dst, Vector _src) {
  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *dst = device.getVector(_dst);
    floatType *src = device.getVector(_src);

    checkedMemcpyAsync(dst, src, sizeof(floatType) * length,
                       hipMemcpyDeviceToDevice);
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::matvecGatherXViaHost(Vector _x) {
  floatType *xHost;
  switch (_x) {
  case VectorX:
    xHost = x;
    break;
  case VectorP:
    xHost = p;
    break;
  default:
    assert(0 && "Invalid vector!");
    return;
  }

  // Gather x on host.
  for (MultiDevice &device : devices) {
    device.setDevice();

    int offset = workDistribution->offsets[device.id];
    int length = workDistribution->lengths[device.id];
    floatType *x = device.getVector(_x);

    checkedMemcpyAsync(xHost + offset, x, sizeof(floatType) * length,
                       hipMemcpyDeviceToHost, device.gatherStream);
  }
  synchronizeAllDevices();

  // Transfer x to devices.
  for (MultiDevice &device : devices) {
    device.setDevice();

    floatType *x = device.getVector(_x, /* forMatvec= */ true);

    for (MultiDevice &src : devices) {
      if (src.id == device.id) {
        // Don't transfer chunk that is already on the device.
        continue;
      }
      int offset = workDistribution->offsets[src.id];
      int length = workDistribution->lengths[src.id];

      checkedMemcpyAsyncToDevice(x + offset, xHost + offset,
                                 sizeof(floatType) * length,
                                 device.gatherStream);
    }
  }
  synchronizeAllDevices();
}

void CGMultiCUDA::matvecGatherXOnDevices(Vector _x) {
  for (MultiDevice &device : devices) {
    device.setDevice();
    floatType *x = device.getVector(_x, /* forMatvec= */ true);

    for (MultiDevice &src : devices) {
      if (src.id == device.id) {
        // Don't transfer chunk that is already on the device.
        continue;
      }

      int offset = workDistribution->offsets[src.id];
      int length = workDistribution->lengths[src.id];
      floatType *xSrc = src.getVector(_x, /* forMatvec= */ true);

      checkedMemcpyAsync(x + offset, xSrc + offset, sizeof(floatType) * length,
                         hipMemcpyDeviceToDevice, device.gatherStream);
    }
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::matvecKernel(Vector _x, Vector _y) {
  switch (gatherImpl) {
  case GatherImplHost:
    matvecGatherXViaHost(_x);
    break;
  case GatherImplDevice:
  case GatherImplP2P:
    matvecGatherXOnDevices(_x);
    break;
  default:
    assert(0 && "Invalid gather implementation!");
  }

  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *x = device.getVector(_x, /* forMatvec= */ true);
    floatType *y = device.getVector(_y);

    switch (matrixFormat) {
    case MatrixFormatCRS:
      matvecKernelCRS<<<device.blocksMatvec, Device::Threads>>>(
          device.matrixCRS.ptr, device.matrixCRS.index, device.matrixCRS.value,
          x, y, length);
      break;
    case MatrixFormatELL:
      matvecKernelELL<<<device.blocksMatvec, Device::Threads>>>(
          device.matrixELL.length, device.matrixELL.index,
          device.matrixELL.data, x, y, length);
      break;
    default:
      assert(0 && "Invalid matrix format!");
    }
    checkLastError();
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::axpyKernel(floatType a, Vector _x, Vector _y) {
  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *x = device.getVector(_x);
    floatType *y = device.getVector(_y);

    axpyKernelCUDA<<<device.blocks, Device::Threads>>>(a, x, y, length);
    checkLastError();
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::xpayKernel(Vector _x, floatType a, Vector _y) {
  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *x = device.getVector(_x);
    floatType *y = device.getVector(_y);

    xpayKernelCUDA<<<device.blocks, Device::Threads>>>(x, a, y, length);
    checkLastError();
  }

  synchronizeAllDevices();
}

floatType CGMultiCUDA::vectorDotKernel(Vector _a, Vector _b) {
  // This is needed for warpReduceSum on __CUDA_ARCH__ < 350
  size_t sharedForVectorDot =
      max(Device::Threads, BlockReduction) * sizeof(floatType);
  size_t sharedForReduce =
      max(Device::MaxBlocks, BlockReduction) * sizeof(floatType);

  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *a = device.getVector(_a);
    floatType *b = device.getVector(_b);

    // https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
    vectorDotKernelCUDA<<<device.blocks, Device::Threads, sharedForVectorDot>>>(
        a, b, device.tmp, length);
    checkLastError();
    deviceReduceKernel<<<1, Device::MaxBlocks, sharedForReduce>>>(
        device.tmp, device.tmp, device.blocks);
    checkLastError();

    checkedMemcpyAsync(&device.vectorDotResult, device.tmp, sizeof(floatType),
                       hipMemcpyDeviceToHost);
  }

  // Synchronize devices and reduce partial results.
  floatType res = 0;
  for (MultiDevice &device : devices) {
    device.setDevice();
    checkedSynchronize();
    res += device.vectorDotResult;
  }

  return res;
}

void CGMultiCUDA::applyPreconditionerKernel(Vector _x, Vector _y) {
  for (MultiDevice &device : devices) {
    device.setDevice();

    int length = workDistribution->lengths[device.id];
    floatType *x = device.getVector(_x);
    floatType *y = device.getVector(_y);

    switch (preconditioner) {
    case PreconditionerJacobi:
      applyPreconditionerKernelJacobi<<<device.blocks, Device::Threads>>>(
          device.jacobi.C, x, y, length);
      break;
    default:
      assert(0 && "Invalid preconditioner!");
    }
    checkLastError();
  }

  synchronizeAllDevices();
}

void CGMultiCUDA::printSummary() {
  CG::printSummary();

  std::cout << std::endl;
  std::string gatherImplName;
  switch (gatherImpl) {
  case GatherImplHost:
    gatherImplName = "via host";
    break;
  case GatherImplDevice:
    gatherImplName = "between devices, but no peer-to-peer";
    break;
  case GatherImplP2P:
    gatherImplName = "peer-to-peer (NVLink)";
    break;
  }
  assert(gatherImplName.length() > 0);
  printPadded("Gather implementation:", gatherImplName);
}

CG *CG::getInstance() { return new CGMultiCUDA; }
